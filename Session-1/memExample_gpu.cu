
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

__global__ void swap_gpu(int *a, int *b)
{
 int tmp = *a;
 *a = *b;
 *b = tmp;
}

int main()
{
 int h_a, h_b;
 h_a = 3;
 h_b = 9;

 int *dev_a, *dev_b;

 size_t varSize = sizeof(int);
 
 hipMalloc((void **)&dev_a, varSize);
 hipMalloc((void **)&dev_b, varSize); 
 
 hipMemcpy(dev_a, &h_a, varSize, hipMemcpyHostToDevice);
 hipMemcpy(dev_b, &h_b, varSize, hipMemcpyHostToDevice); 
 
 swap_gpu<<<1,1>>>(dev_a,dev_b);
 
 hipMemcpy(&h_a, dev_a, varSize, hipMemcpyDeviceToHost);
 hipMemcpy(&h_b, dev_b, varSize, hipMemcpyDeviceToHost);

 hipDeviceSynchronize();

 assert(h_a == 9);
 assert(h_b == 3);

 hipFree(dev_a);
 hipFree(dev_b);

 return 0;
}
